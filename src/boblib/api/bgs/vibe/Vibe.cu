#include "hip/hip_runtime.h"

#include <opencv2/cudaarithm.hpp>

namespace boblib::bgs
{
    __device__ uint32_t xorshift32(uint32_t *state)
    {
        uint32_t x = *state;
        x ^= x << 13;
        x ^= x >> 17;
        x ^= x << 5;
        *state = x;
        return x;
    }

    // CUDA device function to get neighbor position
    __device__ int get_neighbor_position_3x3(const int x, const int y, const int img_width, const int img_height, uint32_t randIdx)
    {
        const int s_anNeighborPattern[8][2] = {
            {-1, 1},
            {0, 1},
            {1, 1},
            {-1, 0},
            {1, 0},
            {-1, -1},
            {0, -1},
            {1, -1},
        };

        const int r = randIdx & 0x7;
        const int nNeighborCoord_X = max(min(x + s_anNeighborPattern[r][0], img_width - 1), 0);
        const int nNeighborCoord_Y = max(min(y + s_anNeighborPattern[r][1], img_height - 1), 0);

        return (nNeighborCoord_Y * img_width + nNeighborCoord_X);
    }

    // CUDA kernel function
    __global__ void vibeKernel(const uchar *d_image, uchar *d_fg_mask, const uchar *d_detect_mask, uchar **d_bg_img_samples,
                               int img_width, int img_height, int n_color_dist_threshold, int bg_samples,
                               int required_bg_samples, int and_learning_rate, bool has_detect_mask, uint32_t *rand_states)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int tid = y * img_width + x;

        if (x >= img_width || y >= img_height)
            return;

        if (has_detect_mask && d_detect_mask[tid] == 0)
        {
            return;
        }

        uint32_t n_good_samples_count = 0;
        uint32_t n_sample_idx = 0;

        uchar pix_data = d_image[tid];

        // Use a local copy of the RNG state
        uint32_t rand_state = rand_states[tid];

        while (n_sample_idx < bg_samples)
        {
            if (abs(static_cast<int32_t>(d_bg_img_samples[n_sample_idx][tid]) - static_cast<int32_t>(pix_data)) < n_color_dist_threshold)
            {
                ++n_good_samples_count;
                if (n_good_samples_count >= required_bg_samples)
                {
                    break;
                }
            }
            ++n_sample_idx;
        }

        if (n_good_samples_count < required_bg_samples)
        {
            d_fg_mask[tid] = UCHAR_MAX;
        }
        else
        {
            if ((xorshift32(&rand_state) & and_learning_rate) == 0)
            {
                d_bg_img_samples[xorshift32(&rand_state) & (bg_samples - 1)][tid] = pix_data;
            }
            if ((xorshift32(&rand_state) & and_learning_rate) == 0)
            {
                int neigh_data = get_neighbor_position_3x3(x, y, img_width, img_height, xorshift32(&rand_state));
                d_bg_img_samples[xorshift32(&rand_state) & (bg_samples - 1)][neigh_data] = pix_data;
            }
        }

        // Store the updated RNG state
        rand_states[tid] = rand_state;
    }
}